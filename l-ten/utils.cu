#include <assert.h>
#include <cstdint>
#include <stdio.h>
#include "utils.h"
#include "globals.h"
#include "error.h"

void* gpu_alloc(uint64_t size)
{
	void* memory = nullptr;
#ifdef USE_CUDA
	LTEN_CUDA_CHECK(hipMalloc(&memory, size));
#else
	LTEN_ERR("The USE_CUDA flag was not be set during the build (this flag must be set in order to use GPU tensors)");
#endif
	return memory;
}

void gpu_free(void* memory)
{
#ifdef USE_CUDA
	LTEN_CUDA_CHECK(hipFree(memory));
#else
LTEN_ERR("The USE_CUDA flag was not be set during the build (this flag must be set in order to use GPU tensors)");
#endif

}


int AllocateMemoryOnGPU(void** memory_ptr_addr, uint64_t size, bool zero_memory)
{
#ifdef USE_CUDA
#ifdef USE_MEMORYPOOL
	*memory_ptr_addr = lten::MISC_globals::singleton()->get_gpu_memorypool()->AllocateMemory(size);
#else
	*memory_ptr_addr = gpu_alloc(size);
#endif

	if (zero_memory)
	{
		ZeroMemoryOnGPU(*memory_ptr_addr, size);
	}
#else
	LTEN_ERR("The USE_CUDA flag was not be set during the build (this flag must be set in order to use GPU tensors)");
#endif
	return 0;
}


void ZeroMemoryOnGPU(void* memory, size_t size)
{
#ifdef USE_CUDA
	LTEN_CUDA_CHECK(hipMemset(memory, 0, size));
#else
LTEN_ERR("The USE_CUDA flag was not be set during the build (this flag must be set in order to use GPU tensors)");
#endif
}

void FreeMemoryOnGPU(void* memory)
{
	if (!memory)
	{
		return;
	}

#ifdef USE_CUDA
#ifdef USE_MEMORYPOOL
	lten::MISC_globals::singleton()->get_gpu_memorypool()->FreeMemory(memory);
#else
	gpu_free(memory);
#endif
#else
	LTEN_ERR("The USE_CUDA flag was not be set during the build (this flag must be set in order to use GPU tensors)");
#endif
}


int CopyDataToGPU(void* gpu, void* host, size_t size)
{
#ifdef USE_CUDA
	LTEN_CUDA_CHECK(hipMemcpy(gpu, host, size, hipMemcpyHostToDevice));
#else
LTEN_ERR("The USE_CUDA flag was not be set during the build (this flag must be set in order to use GPU tensors)");
#endif
	return 0;
}



int CopyDataFromGPU(void* host, void* gpu, size_t size)
{
#ifdef USE_CUDA
	LTEN_CUDA_CHECK(hipMemcpy(host, gpu, size, hipMemcpyDeviceToHost));
#else
	LTEN_ERR("The USE_CUDA flag was not be set during the build (this flag must be set in order to use GPU tensors)");
#endif

	return 0;
}


int GPUToGPUCopy(void* dst, void* src, size_t size)
{
#ifdef USE_CUDA
	LTEN_CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice));
#else
LTEN_ERR("The USE_CUDA flag was not be set during the build (this flag must be set in order to use GPU tensors)");
#endif
	return 0;
}


int GetDevice(int* device)
{
#ifdef USE_CUDA
	LTEN_CUDA_CHECK(hipGetDevice(device));
#else
	LTEN_ERR("The USE_CUDA flag was not be set during the build (this flag must be set in order to use GPU tensors)");
#endif
	return 0;
}

#ifdef USE_CUDA
void cudaErrCheck_(hipError_t stat, const char *file, int line) 
{
	if (stat != hipSuccess) 
	{
		fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
	}
}

void cudnnErrCheck_(hipdnnStatus_t stat, const char *file, int line) 
{
	if (stat != HIPDNN_STATUS_SUCCESS) 
	{
		fprintf(stderr, "cuDNN Error: %s %s %d\n", hipdnnGetErrorString(stat), file, line);
	}
}

void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line)
{
	if (stat != HIPBLAS_STATUS_SUCCESS) 
	{
		fprintf(stderr, "cuBlas Error: %d %s %d\n", cublasGetError(), file, line);
	}
}
#endif


