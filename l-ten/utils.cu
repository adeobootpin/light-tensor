#include <assert.h>
#include <cstdint>
#include <stdio.h>
#include "utils.h"
#include "globals.h"
#include "error.h"

#ifdef USE_CUDA
void* gpu_alloc(uint64_t size)
{
	void* memory;

	LTEN_CUDA_CHECK(hipMalloc(&memory, size));

	return memory;
}

void gpu_free(void* memory)
{
	LTEN_CUDA_CHECK(hipFree(memory));
}


int AllocateMemoryOnGPU(void** memory_ptr_addr, uint64_t size, bool zero_memory)
{
#ifdef USE_MEMORYPOOL
	*memory_ptr_addr = lten::MISC_globals::singleton()->get_gpu_memorypool()->AllocateMemory(size);
#else
	*memory_ptr_addr = gpu_alloc(size);
#endif

	if (zero_memory)
	{
		ZeroMemoryOnGPU(*memory_ptr_addr, size);
	}
	return 0;
}


void ZeroMemoryOnGPU(void* memory, size_t size)
{
	LTEN_CUDA_CHECK(hipMemset(memory, 0, size));
}

void FreeMemoryOnGPU(void* memory)
{
#ifdef USE_MEMORYPOOL
	lten::MISC_globals::singleton()->get_gpu_memorypool()->FreeMemory(memory);
#else
	gpu_free(memory);
#endif
}


int CopyDataToGPU(void* gpu, void* host, size_t size)
{
	LTEN_CUDA_CHECK(hipMemcpy(gpu, host, size, hipMemcpyHostToDevice));
	return 0;
}



int CopyDataFromGPU(void* host, void* gpu, size_t size)
{
	
	LTEN_CUDA_CHECK(hipMemcpy(host, gpu, size, hipMemcpyDeviceToHost));

	return 0;
}


int GPUToGPUCopy(void* dst, void* src, size_t size)
{
	LTEN_CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice));
	return 0;
}


int GetDevice(int* device)
{
	LTEN_CUDA_CHECK(hipGetDevice(device));

	return 0;
}

void cudaErrCheck_(hipError_t stat, const char *file, int line) 
{
	if (stat != hipSuccess) 
	{
		fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
	}
}

void cudnnErrCheck_(hipdnnStatus_t stat, const char *file, int line) 
{
	if (stat != HIPDNN_STATUS_SUCCESS) 
	{
		fprintf(stderr, "cuDNN Error: %s %s %d\n", hipdnnGetErrorString(stat), file, line);
	}
}

void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line)
{
	if (stat != HIPBLAS_STATUS_SUCCESS) 
	{
		fprintf(stderr, "cuBlas Error: %d %s %d\n", cublasGetError(), file, line);
	}
}
#endif



